#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include "xor_wow_data.hpp"
#include "RNGState.h"

struct RNG
{
	unsigned* d_sequence_matrix;
	unsigned* d_offset_matrix;

	__device__ inline void state_init(unsigned long long seed,
		unsigned long long subsequence,
		unsigned long long offset,
		RNGState& state)
	{
		unsigned int s0 = ((unsigned int)seed) ^ 0xaad26b49UL;
		unsigned int s1 = (unsigned int)(seed >> 32) ^ 0xf7dcefddUL;
		unsigned int t0 = 1099087573UL * s0;
		unsigned int t1 = 2591861531UL * s1;
		state.d = 6615241 + t1 + t0;
		state.v.v0 = 123456789UL + t0;
		state.v.v1 = 362436069UL ^ t0;
		state.v.v2 = 521288629UL + t1;
		state.v.v3 = 88675123UL ^ t1;
		state.v.v4 = 5783321UL + t0;

		// apply sequence matrix
		V5 result;
		unsigned long long p = subsequence;
		int i_mat = 0;
		unsigned matrix[800];
		unsigned matrixA[800];

		while (p && i_mat < 7)
		{
			for (unsigned int t = 0; t < (p & 3); t++)
			{
				matvec(state.v, d_sequence_matrix + i_mat * 800, result);
				state.v = result;
			}
			p >>= 2;
			i_mat++;
		}
		if (p)
		{
			memcpy(matrix, d_sequence_matrix + i_mat * 800, sizeof(unsigned) * 800);
			memcpy(matrixA, d_sequence_matrix + i_mat * 800, sizeof(unsigned) * 800);
		}

		while (p)
		{
			for (unsigned int t = 0; t < (p & 0xF); t++)
			{
				matvec(state.v, matrixA, result);
				state.v = result;
			}
			p >>= 4;
			if (p)
			{
				for (int i = 0; i < 4; i++)
				{
					matmat(matrix, matrixA);
					memcpy(matrixA, matrix, sizeof(unsigned) * 800);
				}
			}
		}

		// apply offset matrix
		p = offset;
		i_mat = 0;
		while (p && i_mat < 7)
		{
			for (unsigned int t = 0; t < (p & 3); t++)
			{
				matvec(state.v, d_offset_matrix + i_mat * 800, result);
				state.v = result;
			}
			p >>= 2;
			i_mat++;
		}

		if (p)
		{
			memcpy(matrix, d_offset_matrix + i_mat * 800, sizeof(unsigned) * 800);
			memcpy(matrixA, d_offset_matrix + i_mat * 800, sizeof(unsigned) * 800);
		}

		while (p)
		{

			for (unsigned int t = 0; t < (p & 0xF); t++)
			{
				matvec(state.v, matrixA, result);
				state.v = result;
			}
			p >>= 4;
			if (p)
			{
				for (int i = 0; i < 4; i++)
				{
					matmat(matrix, matrixA);
					memcpy(matrixA, matrix, sizeof(unsigned) * 800);
				}
			}
		}
		state.d += 362437 * (unsigned int)offset;
	}

private:
	static __device__ inline void matvec_i(int i, unsigned v_i, const unsigned *matrix, V5& result)
	{
		for (int j = 0; j < 32; j++)
			if (v_i & (1 << j))
			{
				V5 mat_row = ((V5*)matrix)[i * 32 + j];
				result.v0 ^= mat_row.v0;
				result.v1 ^= mat_row.v1;
				result.v2 ^= mat_row.v2;
				result.v3 ^= mat_row.v3;
				result.v4 ^= mat_row.v4;
			}
	}

	static __device__ inline void matvec(const V5& vector, const unsigned *matrix, V5& result)
	{
		memset(&result, 0, sizeof(V5));
		matvec_i(0, vector.v0, matrix, result);
		matvec_i(1, vector.v1, matrix, result);
		matvec_i(2, vector.v2, matrix, result);
		matvec_i(3, vector.v3, matrix, result);
		matvec_i(4, vector.v4, matrix, result);
	}

	static __device__ inline void matmat(unsigned int *matrixA, const unsigned int *matrixB)
	{
		V5 result;
		for (int i = 0; i < 160; i++)
		{
			matvec(((V5*)matrixA)[i], matrixB, result);
			((V5*)matrixA)[i] = result;
		}
	}
};

__global__
void g_rand_init(RNG rng, RNGState* d_states, unsigned count)
{
	unsigned id = threadIdx.x + blockIdx.x*blockDim.x;
	if (id >= count) return;
	rng.state_init(1234, id, 0, d_states[id]);
}

void cu_rand_init(unsigned count, RNGState* d_states)
{
	RNG rng;
	hipMalloc(&rng.d_sequence_matrix, sizeof(unsigned) * 800 * 8);
	hipMalloc(&rng.d_offset_matrix, sizeof(unsigned) * 800 * 8);
	hipMemcpy(rng.d_sequence_matrix, xorwow_sequence_matrix, sizeof(unsigned) * 800 * 8, hipMemcpyHostToDevice);
	hipMemcpy(rng.d_offset_matrix, xorwow_offset_matrix, sizeof(unsigned) * 800 * 8, hipMemcpyHostToDevice);

	unsigned blocks = (count + 127) / 128;
	g_rand_init << < blocks, 128 >> > (rng, d_states, count);

	hipFree(rng.d_offset_matrix);
	hipFree(rng.d_sequence_matrix);

}

void h_rand_init(unsigned count, RNGState* h_states)
{

	RNGState* d_states;
	hipMalloc(&d_states, sizeof(RNGState)* count);	

	cu_rand_init(count, d_states);

	hipMemcpy(h_states, d_states, sizeof(RNGState)* count, hipMemcpyDeviceToHost);
	hipFree(d_states);
}
