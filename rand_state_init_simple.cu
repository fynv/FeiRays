#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include "RNGState_simple.h"

__global__
void g_rand_init(RNGState* d_states, unsigned width, unsigned height)
{
	unsigned x = threadIdx.x + blockIdx.x*blockDim.x;
	unsigned y = threadIdx.y + blockIdx.y*blockDim.y;
	if (x >= width || y>=height) return;
	unsigned v0 = x, v1 = y, s0 = 0;
	for (unsigned n = 0; n < 16; n++)
	{
		s0 += 0x9e3779b9;
		v0 += ((v1 << 4) + 0xa341316c) ^ (v1 + s0) ^ ((v1 >> 5) + 0xc8013ea4);
		v1 += ((v0 << 4) + 0xad90777d) ^ (v0 + s0) ^ ((v0 >> 5) + 0x7e95761e);
	}
	d_states[x + y * width] = v0;
}


void cu_rand_init(unsigned width, unsigned height, RNGState* d_states)
{
	dim3 dimBlock, dimGrid;
	dimBlock.x = 8;
	dimBlock.y = 8;
	dimGrid.x = (width + 7) / 8;
	dimGrid.y = (height + 7) / 8;
	g_rand_init << < dimBlock, dimGrid >> > ( d_states, width, height);
}

void h_rand_init(unsigned width, unsigned height, RNGState* h_states)
{
	RNGState* d_states;
	hipMalloc(&d_states, sizeof(RNGState)* width*height);

	cu_rand_init(width, height, d_states);

	hipMemcpy(h_states, d_states, sizeof(RNGState)* width*height, hipMemcpyDeviceToHost);
	hipFree(d_states);
}

