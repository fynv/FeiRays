#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <memory.h>
#include <math.h>

inline __device__ float d_clamp01(float f)
{
	float v = f;
	if (v < 0.0f) v = 0.0f;
	else if (v > 1.0f) v = 1.0f;
	return v;
}

__global__
void g_raw_to_srgb(const float* raw, unsigned char* srgb, size_t num_pixels, float boost)
{
	size_t pix_id = threadIdx.x + blockIdx.x*blockDim.x;
	if (pix_id < num_pixels)
	{
		const float* pIn = raw + pix_id * 4;
		unsigned char* pOut = srgb + pix_id * 3;
		float4 v;
		float power = 1.0f / 2.2f;
		v.x = d_clamp01(powf(pIn[0] * boost, power));
		v.y = d_clamp01(powf(pIn[1] * boost, power));
		v.z = d_clamp01(powf(pIn[2] * boost, power));

		pOut[0] = v.x *255.0f + 0.5f;
		pOut[1] = v.y *255.0f + 0.5f;
		pOut[2] = v.z *255.0f + 0.5f;
	}
}

void h_raw_to_srgb(const float* raw, unsigned char* srgb, size_t num_pixels, float boost)
{
	unsigned num_blocks = (unsigned)((num_pixels + 127) / 128);
	g_raw_to_srgb << <num_blocks, 128 >> > (raw, srgb, num_pixels, boost);
}

